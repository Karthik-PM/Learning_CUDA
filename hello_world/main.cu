
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void greet(){
    printf("Hello, World\n");
}

int main(){
    greet<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
