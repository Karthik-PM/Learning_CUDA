#include <bits/stdc++.h>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <ctime>
__global__ void vectorAddition(int* V1, int* V2, int* output, int count){
    int id = blockIdx.x * blockDim.x + threadIdx.x; 

    if(id < count){
        output[id] = V1[id] + V2[id];
    }
}

void display(int * A, int count){
    for(int i = 0; i < count; i++){
        std::cout << A[i] << "\n";
    }
}
int main(){
    srand(time(NULL));
    int count = 1000;
    int *h_a, *h_b, *h_output;

    // Allocating space in CPU space
    h_a = (int*) malloc(sizeof(int) * count);
    h_b = (int*) malloc(sizeof(int) * count);
    h_output = (int*) malloc(sizeof(int) * count);

    // Assigning input array or vectors
    for(int i = 0; i < count; i++){
        h_a[i] = rand() % 1000;
        h_b[i] = rand() % 1000;
    } 
    int *d_a, *d_b, *d_output;
    
    // Allocating memory in GPU space
    if(hipMalloc(&d_a, sizeof(int) * count) != hipSuccess){
        std::cout << "Error Allocating Memory In GPU\n";
    }

    if(hipMalloc(&d_b, sizeof(int) * count) != hipSuccess){
        std::cout << "Error Allcating Memory In GPU\n";
    }
    
    if(hipMalloc(&d_output, sizeof(int) * count) != hipSuccess){
        std::cout << "Error Allcating Memory In GPU\n";
    }

    // copying input contents from CPU to GPU
    if(hipMemcpy(d_a, h_a, sizeof(int) * count, hipMemcpyHostToDevice) != hipSuccess){
        std::cout << "Error copying values from CPU to GPU\n";
    }

    if(hipMemcpy(d_b, h_b, sizeof(int) * count, hipMemcpyHostToDevice) != hipSuccess){
        std::cout << "Error copying values from CPU to GPU\n";
    }
    // copying result to CPU from GPU
    vectorAddition<<<count / 256 + 1, 256>>>(d_a, d_b, d_output, count);
    if(hipMemcpy(h_output, d_output, sizeof(int) * count, hipMemcpyDeviceToHost) != hipSuccess){
        std::cout << "Error copying result from GPU to CPU memory\n";
    }
    display(h_output, count);
    // displaying result
    free(h_a);
    free(h_b);
    free(h_output);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_output);
    return 0;
}
