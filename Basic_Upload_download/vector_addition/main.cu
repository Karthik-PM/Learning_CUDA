
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <stdio.h>
__global__ void vectorAdder(int* A, int* B, int* output, int N){
    for(int i = 0; i < N; i++){
        output[i] = A[i] + B[i];
    }
}

int main(){
    

    // Vectors in CPU
    int *V1, *V2, *output;
    
    // Computed Vector in GPU
    int *V1_GPU, *V2_GPU, *output_GPU;

    int N = 3; // Memory space needed

    // allocating memory in CPU
    V1 = (int*) malloc(sizeof(int) * N);
    V2 = (int*) malloc(sizeof(int) * N);
    output = (int*) malloc(sizeof(int) * N);

    // populating values in the vector
    for(int i = 0; i < N; i++) V1[i] = i;
    for(int i = 0; i < N; i++) V2[i] = i + N;

    // allocating memory for GPU usage
    hipMalloc((void**) &V1_GPU, sizeof(int) * N);
    hipMalloc((void**) &V2_GPU, sizeof(int) * N);
    hipMalloc((void**) &output_GPU, sizeof(int) * N);
    
    // copying the contents of CPU array to GPU array
    hipMemcpy(V1_GPU, V1, sizeof(int) * N, hipMemcpyHostToDevice); // host is descibed as the CPU
    hipMemcpy(V2_GPU, V2, sizeof(int) * N, hipMemcpyHostToDevice);
    
    // host invoking the Device or Kernel (GPU)
    vectorAdder<<<1, 1>>>(V1_GPU, V2_GPU, output_GPU, N);
    hipMemcpy(output, output_GPU, sizeof(int) * N, hipMemcpyDeviceToHost);
    
    // display result
    for(int i = 0; i < N; i++){
        printf("%d ", output[i]);
    }

    // free GPU memory
    hipFree(V1_GPU);
    hipFree(V2_GPU);
    hipFree(output_GPU);

    // free CPU memory
    free(V1);
    free(V2);
    free(output);
    return 0;
}
